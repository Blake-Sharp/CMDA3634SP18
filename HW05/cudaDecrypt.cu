#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ int modprodcu(unsigned int a, unsigned int b, unsigned int p){
	unsigned int za = a;
	unsigned int ab = 0;

	while (b>0){
		if(b%2 ==1) ab = (ab+za)%p;
		za = (2*za)%p;
		b /= 2;
	}
	return ab;
}

__device__ int modExpcu(unsigned int a, unsigned int b, unsigned int p){
	unsigned int z = a;
	unsigned int aExpb = 1;

	while(b>0){
		if(b%2==1) aExpb = modprodcu(aExpb,z,p);
		z = modprodcu(z,z,p);
		b /=2;
	}
	return aExpb;
}
	


//Thought I needed this function... turns out I didnt
//__device__ int parallelFind(unsigned int initialP, unsigned int numLoops){
//
  //   	for (unsigned int i=initialP;i<(intialP+numLoops);i++) {
   //   		if (modExp(g,i+1,p)==h) {
        		//printf("Secret key found! x = %u \n", i+1);
     //   		x=i+1;
//			return x;
//		}
//	} 
//
//}


__global__ void findKey(unsigned int g, unsigned int h, unsigned int p, unsigned int *d_x){ 

	int threadid =threadIdx.x;
	int blockid = blockIdx.x;
	int Nblock = blockDim.x;


	int id = threadid + blockid*Nblock;
	
 
	if (id < (p-1)) {
      		if (modExpcu(g,id+1,p)==h) {
        		//printf("Secret key found! x = %u \n", id+1);
        		*d_x =id+1;
		}
	}
}





int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

	FILE *file;
	file = fopen("public_key.txt","r");
	fscanf(file, "%u %u %u %u", &n,&p,&g,&h);
	fclose(file);

FILE *efile;
efile = fopen("message.txt","r");
fscanf(efile,"%u",&Nints);

  unsigned int *Zmessage =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));

for(unsigned int i = 0; i < Nints; i++){
	fscanf(efile,"%u %u",&(Zmessage[i]),&(a[i]));
}
fclose(efile);


unsigned int h_x;

//	h_g = (unsigned int *) malloc(sizeof(unsigned int));
//	h_h = (unsigned int *) malloc(sizeof(unsigned int));
//	h_x = (unsigned int *) malloc(sizeof(unsigned int));
//	h_p = (unsigned int *) malloc(sizeof(unsigned int));

//	h_g = g;
//	h_h = h;
	h_x = 0;
//	h_p = p;

unsigned int *d_x;

//	hipMalloc(&d_g,sizeof(unsigned int));
//	hipMalloc(&d_h,sizeof(unsigned int));
	hipMalloc(&d_x,sizeof(unsigned int));
//	hipMalloc(&d_p,sizeof(unsigned int));

//	cudeMemcpy(d_g,h_g,sizeof(unsigned int),hipMemcpyHostToDevice)
//	cudeMemcpy(d_h,h_h,sizeof(unsigned int),hipMemcpyHostToDevice)
//	hipMemcpy(d_x,h_x,sizeof(unsigned int),hipMemcpyHostToDevice);
//	cudeMemcpy(d_p,h_p,sizeof(unsigned int),hipMemcpyHostToDevice)

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
	}
    double startTime = clock();
    //for (unsigned int i=0;i<p-1;i++) {
      //if (modExp(g,i+1,p)==h) {
        //printf("Secret key found! x = %u \n", i+1);
        //x=i+1;
	//}
	//}

unsigned int Nthreads = 32;
unsigned int Nblocks = (p-1)/Nthreads;

findKey <<<Nblocks,Nthreads>>>(g,h,p,d_x);
cudeDeviceSynchronize();

hipMemcpy(&h_x,d_x,sizeof(unsigned int),hipMemcpyDeviceToHost);

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

	printf("Secret key found! x = %u \n",h_x);
	printf("It took %d seconds to find the key in cuda.\n", totalTime);
	printf("The work was %d and the throughput was %d.\n",work,throughput);

//hipFree(d_g);
//hipFree(d_p);
//hipFree(d_h);
hipFree(d_x);

//free(h_g);
//free(g_h);
//free(h_x);
//free(h_p);


  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
