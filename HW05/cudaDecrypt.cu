#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ int modprodcu(unsigned int a, unsigned int b, unsigned int p){
	unsigned int za = a;
	unsigned int ab = 0;

	while (b>0){
		if(b%2 ==1) ab = (ab+za)%p;
		za = (2*za)%p;
		b /= 2;
	}
	return ab;
}

__device__ int modExpcu(unsigned int a, unsigned int b, unsigned int p){
	unsigned int z = a;
	unsigned int aExpb = 1;

	while(b>0){
		if(b%2==1) aExpb = modprodcu(aExpb,z,p);
		z = modprodcu(z,z,p);
		b /=2;
	}
	return aExpb;
}
	


//Thought I needed this function... turns out I didnt
//__device__ int parallelFind(unsigned int initialP, unsigned int numLoops){
//
  //   	for (unsigned int i=initialP;i<(intialP+numLoops);i++) {
   //   		if (modExp(g,i+1,p)==h) {
        		//printf("Secret key found! x = %u \n", i+1);
     //   		x=i+1;
//			return x;
//		}
//	} 
//
//}


//<<<<<<< HEAD
// __global__ void findKey(unsigned int g, unsigned int h,unsigned int *d_x unsigned int p){ 
//=======
__global__ void findKey(unsigned int g, unsigned int h, unsigned int p, unsigned int *d_x){ 
//>>>>>>> 080b1748334d1d58f0c74e8f5a99719d01a148fc

	int threadid =threadIdx.x;
	int blockid = blockIdx.x;
	int Nblock = blockDim.x;


	unsigned int id = threadid + blockid*Nblock;
	
 
	if (id < (p-1)) {
      		if (modExpcu(g,id+1,p)==h) {
//<<<<<<< HEAD
//        		printf("Secret key found! x = %u \n", id+1);
//        		d_x=id+1;
//=======
        		//printf("Secret key found! x = %u \n", id+1);
        		*d_x =id+1;
//>>>>>>> 080b1748334d1d58f0c74e8f5a99719d01a148fc
		}
	}
}





int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

	FILE *file;
	file = fopen("bonus_public_key.txt","r");
	fscanf(file, "%u %u %u %u", &n,&p,&g,&h);
	fclose(file);

FILE *efile;
efile = fopen("bonus_message.txt","r");
fscanf(efile,"%u",&Nints);

  unsigned int *Zmessage =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a =
      (unsigned int *) malloc(Nints*sizeof(unsigned int));

for(unsigned int i = 0; i < Nints; i++){
	fscanf(efile,"%u %u",&(Zmessage[i]),&(a[i]));
}
fclose(efile);


unsigned int h_x;

//	h_g = (unsigned int *) malloc(sizeof(unsigned int));
//	h_h = (unsigned int *) malloc(sizeof(unsigned int));
//	h_x = (unsigned int *) malloc(sizeof(unsigned int));
//	h_p = (unsigned int *) malloc(sizeof(unsigned int));

//	h_g = g;
//	h_h = h;
	h_x = 0;
//	h_p = p;

unsigned int *d_x;

//	hipMalloc(&d_g,sizeof(unsigned int));
//	hipMalloc(&d_h,sizeof(unsigned int));
	hipMalloc(&d_x,sizeof(unsigned int));
//	hipMalloc(&d_p,sizeof(unsigned int));

//<<<<<<< HEAD
//	hipMemcpy(d_g,h_g,sizeof(unsigned int),hipMemcpyHostToDevice)
//	hipMemcpy(d_h,h_h,sizeof(unsigned int),hipMemcpyHostToDevice)
//	hipMemcpy(*d_x,h_x,sizeof(unsigned int),hipMemcpyHostToDevice);
//	hipMemcpy(d_p,h_p,sizeof(unsigned int),hipMemcpyHostToDevice)
//=======
//	cudeMemcpy(d_g,h_g,sizeof(unsigned int),hipMemcpyHostToDevice)
//	cudeMemcpy(d_h,h_h,sizeof(unsigned int),hipMemcpyHostToDevice)
//	hipMemcpy(d_x,h_x,sizeof(unsigned int),hipMemcpyHostToDevice);
//	cudeMemcpy(d_p,h_p,sizeof(unsigned int),hipMemcpyHostToDevice)
//>>>>>>> 080b1748334d1d58f0c74e8f5a99719d01a148fc

  // find the secret key
//<<<<<<< HEAD
  if (x==0 || modExp(g,x,p)!=h){
    printf("Finding the secret key...\n");
	}
//=======
  //if (x==0 || modExp(g,x,p)!=h) {
    //printf("Finding the secret key...\n");
//	}
//>>>>>>> 043ba2a664734012364823d7453a549526f335e3
    double startTime = clock();
    //for (unsigned int i=0;i<p-1;i++) {
      //if (modExp(g,i+1,p)==h) {
        //printf("Secret key found! x = %u \n", i+1);
        //x=i+1;
	//}
	//}

unsigned int Nthreads = 32;
unsigned int Nblocks = (p-1)/Nthreads;

//<<<<<<< HEAD
//findKey <<<Nblocks,Nthreads>>>(g,h,*d_x,p);
//hipDeviceSynchronize();

//hipMemcpy(h_x,*d_x,sizeof(unsigned int),hipMemcpyDeviceToHost);
//=======
findKey <<<Nblocks,Nthreads>>>(g,h,p,d_x);
hipDeviceSynchronize();

hipMemcpy(&h_x,d_x,sizeof(unsigned int),hipMemcpyDeviceToHost);
//>>>>>>> 080b1748334d1d58f0c74e8f5a99719d01a148fc

    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

	printf("Secret key found! x = %u \n",h_x);
	printf("It took %d seconds to find the key in cuda.\n", totalTime);
	printf("The work was %d and the throughput was %d.\n",work,throughput);

//hipFree(d_g);
//hipFree(d_p);
//hipFree(d_h);
hipFree(d_x);

//free(h_g);
//free(g_h);
//free(h_x);
//free(h_p);


  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}

